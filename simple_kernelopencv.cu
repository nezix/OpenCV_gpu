#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv_modules.hpp>

//if defined(HAVE_OPENCV_GPU)

#include <opencv2/opencv.hpp>
#include <opencv2/core/opengl_interop.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/contrib/contrib.hpp>

#define RESIZEF 0.5

using namespace std;
using namespace cv;


#include "kernel.h"

//nvcc kernel.cu testgpu.cu -o mygpu.ex `pkg-config --cflags --libs opencv`


uchar3 value_red[2] = {make_uchar3(170,110,90),make_uchar3(179,255,255)};
uchar3 value_blue[2] = {make_uchar3(100,120,50),make_uchar3(130,255,255)};
uchar3 value_green[2] =  {make_uchar3(35,120,50),make_uchar3(80,255,255)};
uchar3 value_yellow[2] = {make_uchar3(20,120,70),make_uchar3(35,255,255)};
uchar3 value_white[2] = { make_uchar3(0,0,150),make_uchar3(179,30,255)};
uchar3 value_black[2] = {make_uchar3(0,0,0),make_uchar3(179,225,70)};


void usage(int argc,char **argv){
    if(argc != 2){
        cout << "Usage : "<<argv[0]<<" namefile"<<endl;
        exit(-1);
    }
}

int main(int argc, char** argv)
{
    usage(argc,argv);
    setUseOptimized(true);

    //namedWindow("GPU",WINDOW_OPENGL);

    gpu::setGlDevice();

    Mat frame;
    VideoCapture capture(argv[1]);
    int counter = 0;

    if(!capture.isOpened()){
        cerr<<"Error while reading file";
        exit(-1);
    }
    while(true){
        capture>>frame;
        if(frame.empty())
            break;
        gpu::GpuMat gpuframe(frame);
        gpu::GpuMat resized;
        gpu::GpuMat hsv;
        gpu::resize(gpuframe, resized, Size(0,0),RESIZEF,RESIZEF);
        gpu::GpuMat mask_red(resized.rows,resized.cols,resized.type());
        
        //mask_red.setTo(Scalar::all(0));
        //if(counter>=50){
            
            gpu::cvtColor(resized,hsv,CV_BGR2HSV);
            
            mygpuinrange(resized.cols,resized.rows,gpu::PtrStep<uchar3>(resized),
                            gpu::PtrStep<uchar3>(mask_red),value_red[0],value_red[1]);

            Mat resframe(mask_red);
        //}

        imshow("GPU",resframe);
        counter++;
        if(waitKey(1)>0 )
            break;
    }

    return 0;
}

//#endif
