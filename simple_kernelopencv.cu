#include "hip/hip_runtime.h"
/* <Written by Xavier Martinez : Shows how to manage CUDA kernels without using hipMalloc >
    Copyright (C) <2014>  <Xavier Martinez>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include <iostream>
#include <opencv2/opencv_modules.hpp>

//if defined(HAVE_OPENCV_GPU)

#include <opencv2/opencv.hpp>
#include <opencv2/core/opengl_interop.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/contrib/contrib.hpp>

#define RESIZEF 0.5

using namespace std;
using namespace cv;


#include "kernel.h"

//nvcc kernel.cu testgpu.cu -o mygpu.ex `pkg-config --cflags --libs opencv`


uchar3 value_red[2] = {make_uchar3(170,110,90),make_uchar3(179,255,255)};
uchar3 value_blue[2] = {make_uchar3(100,120,50),make_uchar3(130,255,255)};
uchar3 value_green[2] =  {make_uchar3(35,120,50),make_uchar3(80,255,255)};
uchar3 value_yellow[2] = {make_uchar3(20,120,70),make_uchar3(35,255,255)};
uchar3 value_white[2] = { make_uchar3(0,0,150),make_uchar3(179,30,255)};
uchar3 value_black[2] = {make_uchar3(0,0,0),make_uchar3(179,225,70)};


void usage(int argc,char **argv){
    if(argc != 2){
        cout << "Usage : "<<argv[0]<<" namefile"<<endl;
        exit(-1);
    }
}

int main(int argc, char** argv)
{
    usage(argc,argv);
    setUseOptimized(true);

    //namedWindow("GPU",WINDOW_OPENGL);

    gpu::setGlDevice();

    Mat frame;
    VideoCapture capture(argv[1]);
    int counter = 0;

    if(!capture.isOpened()){
        cerr<<"Error while reading file";
        exit(-1);
    }
    while(true){
        capture>>frame;
        if(frame.empty())
            break;
        gpu::GpuMat gpuframe(frame);
        gpu::GpuMat resized;
        gpu::GpuMat hsv;
        gpu::resize(gpuframe, resized, Size(0,0),RESIZEF,RESIZEF);
        gpu::GpuMat mask_red(resized.rows,resized.cols,resized.type());
        
        //mask_red.setTo(Scalar::all(0));
        //if(counter>=50){
            
            gpu::cvtColor(resized,hsv,CV_BGR2HSV);
            
            mygpuinrange(resized.cols,resized.rows,gpu::PtrStep<uchar3>(resized),
                            gpu::PtrStep<uchar3>(mask_red),value_red[0],value_red[1]);

            Mat resframe(mask_red);
        //}

        imshow("GPU",resframe);
        counter++;
        if(waitKey(1)>0 )
            break;
    }

    return 0;
}

//#endif
