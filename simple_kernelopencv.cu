#include "hip/hip_runtime.h"
/* <Written by Xavier Martinez : Shows how to manage CUDA kernels without using hipMalloc >
    Copyright (C) <2014>  <Xavier Martinez>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To do : Asynchronous version

#include <iostream>
#include <opencv2/opencv_modules.hpp>

//if defined(HAVE_OPENCV_GPU)

#include <opencv2/opencv.hpp>
#include <opencv2/core/opengl_interop.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/contrib/contrib.hpp>

#define RESIZEF 0.5

using namespace std;
using namespace cv;


#include "kernel.h"

//nvcc kernel.cu simple_kernelopencv.cu -o mygpu.ex `pkg-config --cflags --libs opencv` -arch=sm_13


uchar3 value_red[2] = {make_uchar3(170,110,90),make_uchar3(179,255,255)};


void usage(int argc,char **argv){
    if(argc != 2){
        cout << "Usage : "<<argv[0]<<" namefile"<<endl;
        exit(-1);
    }
}

int main(int argc, char** argv)
{
    usage(argc,argv);
    setUseOptimized(true);

    gpu::setGlDevice();

    //Get the video on GPU (gpu video reader)
    gpu::GpuMat frame;
    gpu::VideoReader_GPU d_reader(argv[1]);
    gpu::GpuMat resized;
    gpu::GpuMat hsv;
    int counter = 0;

    //Gpu matrix allocation
    gpu::GpuMat mask_red(0,0,CV_8U);

    while(true){
        //Read frame
        if (!d_reader.read(frame))
            break;

        //Resize the image on the GPU
        gpu::resize(frame, resized, Size(0,0),RESIZEF,RESIZEF);

        //We use uchar3 not uchar4 frame
        gpu::GpuMat resized2;
        gpu::cvtColor( resized,resized2, CV_BGRA2BGR );

        //Allocate GPU memory if necessary
        gpu::ensureSizeIsEnough(resized.rows,resized.cols,CV_8U,mask_red);  

        //Convert the RGB image in HSV values
        gpu::cvtColor(resized,hsv,CV_BGR2HSV);
        
        //Launch CUDA kernel
        mygpuinrange(hsv,mask_red,value_red[0],value_red[1]);

        //Convert back to RGB
        gpu::GpuMat maskrgb(mask_red);        
        gpu::cvtColor( mask_red, maskrgb, CV_GRAY2BGR );

        //Compute the RGB frame with only red mask
	    gpu::GpuMat result;
        gpu::bitwise_and(resized2,maskrgb,result);

        //Get image from GPU to CPU
        Mat resframe(result);

        imshow("GPU",resframe);
        counter++;
        if(waitKey(30)>0 )
            break;
    }

    return 0;
}

//#endif
