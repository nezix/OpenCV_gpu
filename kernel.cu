#include "hip/hip_runtime.h"
/* <Written by Xavier Martinez : Shows how to manage CUDA kernels without using hipMalloc >
    Copyright (C) <2014>  <Xavier Martinez>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#define BLOCKSIZE 256
#include <iostream>

#include <opencv2/opencv_modules.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/opengl_interop.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/contrib/contrib.hpp>

using namespace std;
using namespace cv;

#include "kernel.h"


void mygpuinrange(gpu::GpuMat &src,gpu::GpuMat &dst,uchar3 min, uchar3 max){
    dim3 threadsPerBlock(BLOCKSIZE,BLOCKSIZE);
    int nbblocks1 = (int)ceil((float)src.cols/threadsPerBlock.x);
    int nbblocks2 = (int)ceil((float)src.rows/threadsPerBlock.y);
    dim3 numblocks(nbblocks1,nbblocks2);

	GpuinRange<<<threadsPerBlock,numblocks>>>(gpu::PtrStep<uchar3>(src),gpu::PtrStep<uchar>(dst),src.cols,src.rows,min,max);
}

__global__ void GpuinRange(gpu::PtrStep<uchar3> src,gpu::PtrStep<uchar> dst,int width,int height,uchar3 valmin, uchar3 valmax){

    int x = (blockIdx.x * blockDim.x) + (threadIdx.x);
    int y = (blockIdx.y * blockDim.y) + (threadIdx.y);

    if(x < width && y < height){
        uchar3 myval = src.ptr(y)[x];
        uchar mydest = (uchar)255;//max value
        
        //Out of range min/max
        if(myval.x < valmin.x || myval.x > valmax.x)
            mydest = (uchar)0;
        if(myval.y < valmin.y || myval.y > valmax.y)
            mydest = (uchar)0;
        if(myval.z < valmin.z || myval.z > valmax.z)
            mydest = (uchar)0;
    
        dst.ptr(y)[x] = mydest;
    }
}
