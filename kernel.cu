#include "hip/hip_runtime.h"
/* <Written by Xavier Martinez : Shows how to manage CUDA kernels without using hipMalloc >
    Copyright (C) <2014>  <Xavier Martinez>

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#define BLOCKSIZE 256
#include <iostream>

#include <opencv2/opencv_modules.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/opengl_interop.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/contrib/contrib.hpp>

using namespace std;
using namespace cv;

#include "kernel.h"


void mygpuinrange(int width,int height,gpu::PtrStep<uchar3> src,gpu::PtrStep<uchar3> dst,uchar3 min, uchar3 max){
	dim3 threadsPerBlock(BLOCKSIZE,BLOCKSIZE);
	dim3 numblocks(width/BLOCKSIZE,height/BLOCKSIZE);
	GpuinRange<<<threadsPerBlock,numblocks>>>(src,dst,width,height,min,max);
}

__global__ void GpuinRange(gpu::PtrStep<uchar3> src,gpu::PtrStep<uchar3> dst,int width,int height,uchar3 valmin, uchar3 valmax){

    int x = (blockIdx.x * blockDim.x) + (threadIdx.x);
    int y = (blockIdx.y * blockDim.y) + (threadIdx.y);

    if(x < width && y < height){
        uchar3 myval = src.ptr(y)[x];
        uchar3 mydest = make_uchar3(255,255,255);

        if(myval.x < valmin.x || myval.x > valmax.x)
            mydest = make_uchar3(0,0,0);
        if(myval.y < valmin.y || myval.y > valmax.y)
            mydest = make_uchar3(0,0,0);
        if(myval.z < valmin.z || myval.z > valmax.z)
            mydest = make_uchar3(0,0,0);
    
        dst.ptr(y)[x] = mydest;
    }
}
