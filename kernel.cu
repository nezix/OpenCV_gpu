#include "hip/hip_runtime.h"
#define BLOCKSIZE 256
#include <iostream>

#include <opencv2/opencv_modules.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/opengl_interop.hpp>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/contrib/contrib.hpp>

using namespace std;
using namespace cv;

#include "kernel.h"


void mygpuinrange(int width,int height,gpu::PtrStep<uchar3> src,gpu::PtrStep<uchar3> dst,uchar3 min, uchar3 max){
	dim3 threadsPerBlock(BLOCKSIZE,BLOCKSIZE);
	dim3 numblocks(width/BLOCKSIZE,height/BLOCKSIZE);
	GpuinRange<<<threadsPerBlock,numblocks>>>(src,dst,width,height,min,max);
}

__global__ void GpuinRange(gpu::PtrStep<uchar3> src,gpu::PtrStep<uchar3> dst,int width,int height,uchar3 valmin, uchar3 valmax){

    int x = (blockIdx.x * blockDim.x) + (threadIdx.x);
    int y = (blockIdx.y * blockDim.y) + (threadIdx.y);

    if(x < width && y < height){
        uchar3 myval = src.ptr(y)[x];
        uchar3 mydest = make_uchar3(255,255,255);

        if(myval.x < valmin.x || myval.x > valmax.x)
            mydest = make_uchar3(0,0,0);
        if(myval.y < valmin.y || myval.y > valmax.y)
            mydest = make_uchar3(0,0,0);
        if(myval.z < valmin.z || myval.z > valmax.z)
            mydest = make_uchar3(0,0,0);
    
        dst.ptr(y)[x] = mydest;
    }
}
